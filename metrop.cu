/*
    // ОДНОСПИНОВЫЙ МЕТРОПОЛИС С УЧЕТОМ ТЕРМОДИНАМИЧЕСКОЙ ВЕРОЯТНОСТИ //
    
    * принимает .mfsys, расчитывает матрицу энергий
    * алг метрополиса с учетом термодинамической вероятности

*/

#include <iostream>
#include <fstream>
#include <vector>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <string>
#include <sstream>
#include <chrono>

#include <hip/hip_runtime.h>

#define threads_in_block 1024

static void HandleError(hipError_t err, const char *file, int line)  // проверка на ошибку при опрерации с device-памятью на host
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__ ))

using namespace std;

class Timer
{
    private:
        std::chrono::time_point<std::chrono::steady_clock> m_StartTime;     // время, (теориетически) не зависящее от cpu
        std::chrono::time_point<std::chrono::steady_clock> m_EndTime;
        bool m_bRunning = false;
    
    public:
        void start()
        {
            m_StartTime = std::chrono::steady_clock::now();
            m_bRunning = true;
        }

        void stop()
        {
            m_EndTime = std::chrono::steady_clock::now();
            m_bRunning = false;
        }
    
        double Milliseconds()
        {
            std::chrono::time_point<std::chrono::steady_clock> endTime;
            
            if(m_bRunning)
            {
                endTime = std::chrono::steady_clock::now();
            }
            else
            {
                endTime = m_EndTime;
            }
            
            return std::chrono::duration_cast<std::chrono::milliseconds>(endTime - m_StartTime).count();
        }
    
        double Seconds()
        {
            return Milliseconds() / 1000.0;
        }
};

// создание матрицы энергий из .mfsys файла
__global__ void matrix_create(double* matrixEn, double* x, double* y, double* mx, double* my, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("%i на блоке %i\n", index, blockIdx.x);

    double r, Xij, Yij;
    int i = index / N;
    int j = index % N;
    if (index < N * N)
    {
        if (i < j)
        {
            Xij = x[i] - x[j];
            Yij = y[i] - y[j];
            r = sqrt((double)(Xij * Xij + Yij * Yij));

            matrixEn[index] = ((mx[i] * mx[j] + my[i] * my[j]) / (r * r * r) - 3. * (mx[i] * Xij + my[i] * Yij) * (mx[j] * Xij + my[j] * Yij) / (r * r * r * r * r));
        }
    }
}

// расчет энергия и по строкам и расчет полной энергии системы
__global__ void calc_E_cuda(double* matrixEn, uint8_t* spins, double* E_line, double* E, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index / N;
    int j = index % N;

    // расчет энергии по строкам
    if (index < N * N)  
    {
        if (i < j)
        {
            int si = 1, sj = 1;
            double tmp = .0;

            if (spins[i] == 0) si = -1;     // значения в excel инвертированы
            if (spins[j] == 0) sj = -1;     // поменять * -1 в условиях, если надо

            tmp += matrixEn[index] * si * sj;   

            // printf("index: %d  value: %lf  line: %d  colomn: %d\n", index, matrixEn[index], i, j);

            atomicAdd(&E_line[i], tmp);
            atomicAdd(&E_line[j], tmp);
            atomicAdd(E, tmp);

            // atomicAdd(int* address, int val)
            // atomicAdd(E_line + i, tmp);
        }
    }
}


// чтение параметров из консоли (имя .mfsys файла и т.д.)
void console_param_read(int argc, char** argv, string& filename)
{
    if(argc >= 2)    
    {
        for (int i = 1; i < argc; i++)      // to_string() - неправильная конвертация ()
        {
            if (string(argv[i]).find(".mfsys") != string::npos)
                filename = string(argv[i]);
            else cout << "Неправильный формат файла!\n";
        }
    }
}

// чтение результата из .mfsys
bool input_from_mfsys(const string& filename,vector <double>& vx, vector <double>& vy, vector <double>& vmx, vector <double>& vmy)
{
    fstream file(filename);
    if (!file.is_open())
    {
        cout << "File not found!" << endl;
        return false;
    }
    else
    {
        string s;
        for (file >> s; s != "[parts]"; file >> s)
            continue;
        int i, state;
        double x, y, z, mx, my, mz;
        while (!file.eof()){
            if(!(file >> i >> x >> y >> z >> mx >> my >> mz >> state))
                break;
            vx.push_back(x);
            vy.push_back(y);
            vmx.push_back(mx);
            vmy.push_back(my);
        }
        return true;
    }
}

// вывод для проверки
void print_2dmatrix(double* matrixEn, const int& N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            cout << matrixEn[N*i + j] << ", ";
        cout << endl;
    }
}

int main( int argc, char** argv )
{   
    srand(time (NULL));

    string filename = "points376_80.mfsys";
    console_param_read(argc, argv, filename);     // чтение параметрова из консоли

    vector <double> vx, vy, vmx, vmy;
    if (!input_from_mfsys(filename, vx, vy, vmx, vmy)) return -1;   // если файл не найден

    const int N = vx.size();    // размер системы 
    auto matrixEn = new double[N * N]();    // матрица энергий
    double* dev_matrixEn;
    HANDLE_ERROR(hipMalloc((void**)&dev_matrixEn, sizeof(double) * N * N));
    HANDLE_ERROR(hipMemcpy(dev_matrixEn, matrixEn, sizeof(double) * N * N, hipMemcpyHostToDevice));
    
    double * dev_x, * dev_y, * dev_mx, * dev_my;
    HANDLE_ERROR(hipMalloc((void**)&dev_x, vx.size() * sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_y, vx.size() * sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_mx, vx.size() * sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_my, vx.size() * sizeof(double)));
    HANDLE_ERROR(hipMemcpy(dev_x, vx.data(), vx.size() * sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_y, vy.data(), vx.size() * sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_mx, vmx.data(), vx.size() * sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_my, vmy.data(), vx.size() * sizeof(double), hipMemcpyHostToDevice));

    int blocks = (int)(N * N / threads_in_block) + 1;
    matrix_create <<<blocks,threads_in_block>>> (dev_matrixEn, dev_x, dev_y, dev_mx, dev_my, N);    // создание матрицы энергий

    HANDLE_ERROR(hipMemcpy(matrixEn, dev_matrixEn, sizeof(double) * N * N, hipMemcpyDeviceToHost));

    // print_2dmatrix(matrixEn, N);
    
    uint8_t spins[N];    // Направление суперспинов
    double E_line[N];    // Энергия в каждом ряду
    for (int i = 0; i < N; i++)
    {
        spins[i] = 0;   // все вниз
        E_line[i] = .0;
    }
    // spins[N-1] = 1;

    // столбцы с плюсовой и минусовой энергией
    vector <int> pos_E_ind;
    vector <int> neg_E_ind;


    uint8_t* dev_spins;
    double* dev_E_line;
    HANDLE_ERROR(hipMalloc((void**)&dev_spins, sizeof(uint8_t) * N));
    HANDLE_ERROR(hipMemcpy(dev_spins, spins, sizeof(uint8_t) * N, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMalloc((void**)&dev_E_line, sizeof(double) * N));
    HANDLE_ERROR(hipMemcpy(dev_E_line, E_line, sizeof(double) * N, hipMemcpyHostToDevice));


    // auto mexp = new double [N]; 
    double Z = .0, E = 0, E2 = .0, E_aver = .0, E2_aver = .0;
    double * dev_mexp, * dev_Z, * dev_E, * dev_E2;

    HANDLE_ERROR(hipMemcpy(dev_matrixEn, matrixEn, sizeof(double) * N * N, hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMalloc( (void**)&dev_mexp, sizeof(double) * N));
    HANDLE_ERROR(hipMalloc( (void**)&dev_Z, sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_E, sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_E2, sizeof(double)));

    HANDLE_ERROR(hipMemset(dev_E, 0, sizeof(double)));
    HANDLE_ERROR(hipMemset(dev_E2, 0, sizeof(double)));
    HANDLE_ERROR(hipMemset(dev_Z, 0, sizeof(double)));

    Timer timer;
    timer.start();

    // Перед матрополисом происходит расчет начального E и массивов, чтобы отталкиваться от него на первой итерации
    calc_E_cuda <<<blocks,threads_in_block>>>(dev_matrixEn, dev_spins, dev_E_line, dev_E, N);

    HANDLE_ERROR(hipMemcpy( E_line, dev_E_line, sizeof(double)*N, hipMemcpyDeviceToHost));    // скачивание на cpu
    HANDLE_ERROR(hipMemcpy( &E, dev_E, sizeof(double), hipMemcpyDeviceToHost));

    for (int i = 0; i < N; i++)
    {
        if (E_line[i] > 0)
        {
            pos_E_ind.push_back(i);
        }
        else if (E_line[i] < 0)
        {
            neg_E_ind.push_back(i);
        }
        else if (E_line[i] == 0)
        {
            cout << "Энергия строки равна 0! Вероятнa ошибка!" << endl;
            return - 1;
        }

        cout << i << ": " << E_line[i] << endl;
    }
    cout << endl << "E = " << E << endl;

    cout << "Строки с положительными энергиями: ";
    for (int tmp : pos_E_ind)   cout << tmp << " ";     cout << endl;
    cout << "Строки с отрицательными энергиями: ";
    for (int tmp : neg_E_ind)   cout << tmp << " ";     cout << endl;

    // начало метрополиса
    // for (double T = 0.001; T<4; T+=0.1)
    for (double T = 0.001; T<0.101; T+=0.1)
    {
        E_aver = .0; // средние энергии
        E2_aver = .0;

        // for (int MK = 0; MK < N * 100; MK++)
        for (int MK = 0; MK < 1; MK++)
        {
            E2 = .0;

            int rand_spin = rand() % N;
            if (spins[rand_spin] == 1)  spins[rand_spin] = 0;
            else if (spins[rand_spin] == 0)  spins[rand_spin] = 1;      // случайный спин переворачивается

            for (int i = 0; i < N; i++) cout << unsigned(spins[i]) << ", ";  cout << endl;

            for (int i = 0; i < N; i++)    E_line[i] = .0;

            HANDLE_ERROR(hipMemcpy(dev_spins, spins, sizeof(uint8_t) * N, hipMemcpyHostToDevice));
            HANDLE_ERROR(hipMemcpy(dev_E_line, E_line, sizeof(double) * N, hipMemcpyHostToDevice));
            HANDLE_ERROR(hipMemset(dev_E, 0, sizeof(double)));

            // 1 расчет E
            calc_E_cuda <<<blocks,threads_in_block>>>(dev_matrixEn, dev_spins, dev_E_line, dev_E, N);

            HANDLE_ERROR(hipMemcpy( E_line, dev_E_line, sizeof(double)*N, hipMemcpyDeviceToHost));
            HANDLE_ERROR(hipMemcpy( &E2, dev_E, sizeof(double), hipMemcpyDeviceToHost));

            for (int i = 0; i < N; i++)
            {
                cout << i << ": " << E_line[i] << endl;
            }
            cout << endl << "E2 = " << E2 << endl;

            if (E > E2)
            {
                // переворот в зависимости от теромодинамической вероятности
            }
        }

    }

    timer.stop();
    cout << "Время выполнения: " << timer.Milliseconds() << " мсек" << endl;

    return 0;
}